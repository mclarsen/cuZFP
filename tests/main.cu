#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>

using namespace thrust;
using namespace std;

#define FREXP(x, e) frexp(x, e)
#define LDEXP(x, e) ldexp(x, e)

const int nx = 32;
const int ny = 32;
const int nz = 32;
uint mx = 0;
uint my = 0;
uint mz = 0;
size_t blksize = 0;


//Used to generate rand array in CUDA with Thrust
struct RandGen
{
    RandGen() {}

    __device__ float operator () (const uint idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist(-1.0, 1.0);
        randEng.discard(idx);
        return uniDist(randEng);
    }
};


//*****************************************************************
//testFREXP
//Input:
//max_threads, number of items in in and out arrays
//array of in, of type T
//Output: out array
//*****************************************************************
template<class T>
__global__
void cudaTestFREXP
(
        int max_threads,
        const T *in,
        T *out,
        int *nptr
        )
{
    uint idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < max_threads)
        out[idx] = FREXP(in[idx], &nptr[ idx] );

}

//*****************************************************************
//testLDEXP
//Input:
//max_threads, number of items in in and out arrays
//array of in, of type T
//Output: out array
//*****************************************************************
template<class T>
__global__
void cudaTestLDEXP(
        int max_threads,
        const T *in,
        T *out
        )
{
    uint idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < max_threads)
        out[idx] = LDEXP(in[idx], 10);
}

template<class T>
void testFEXP(
        device_vector<T> &in,
        device_vector<T> &out)
{
    device_vector<int> d_vec_nptr(nx*ny*nz);
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord( start, 0 );
    cudaTestFREXP<T><<<nx*ny, nz>>>(
        nx*ny*nz,
        raw_pointer_cast(in.data()),
        raw_pointer_cast(out.data()),
        raw_pointer_cast(d_vec_nptr.data())
    );
    T sum = reduce(
            out.begin(),
            out.end()
        );

    hipEventRecord( stop, 0 );
    hipEventSynchronize(stop);
    hipEventElapsedTime( &time, start, stop );

    cout << "FREXP sum: " << sum << " in time: " << time << endl;

    hipEventDestroy( start );
    hipEventDestroy( stop );

}

template<class T>
void testLDEXP(
        device_vector<T> &in,
        device_vector<T> &out)
{
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord( start, 0 );

    cudaTestLDEXP<T><<<nx*ny, nz>>>(
        nx*ny*nz,
        raw_pointer_cast(in.data()),
        raw_pointer_cast(out.data())
    );
    T sum = reduce(
                out.begin(),
                out.end()
    );
    hipEventRecord( stop, 0 );
    hipEventSynchronize(stop);
    hipEventElapsedTime( &time, start, stop );
    cout << "LDEXP sum: " << sum << " in time: " << time << endl;

}

int main()
{
    device_vector<double> d_vec_in(nx*ny*nz), d_vec_out(nx*ny*nz);
    host_vector<double> h_vec(nx*ny*nz);

    thrust::counting_iterator<uint> index_sequence_begin(0);
    thrust::transform(
                    index_sequence_begin,
                    index_sequence_begin + nx*ny*nz,
                    d_vec_in.begin(),
                    RandGen());

    testFEXP<double>(d_vec_in, d_vec_out);
    testLDEXP<double>(d_vec_in, d_vec_out);

}
