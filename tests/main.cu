#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>

#define KEPLER 0
#include "ErrorCheck.h"

using namespace thrust;
using namespace std;

#define FREXP(x, e) frexp(x, e)
#define LDEXP(x, e) ldexp(x, e)

const int nx = 256;
const int ny = 256;
const int nz = 256;


//Used to generate rand array in CUDA with Thrust
struct RandGen
{
    RandGen() {}

    __device__ float operator () (const uint idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist(-1.0, 1.0);
        randEng.discard(idx);
        return uniDist(randEng);
    }
};


//*****************************************************************
//testFREXP
//Input:
//max_threads, number of items in in and out arrays
//array of in, of type T
//Output: out array
//*****************************************************************
template<class T>
__global__
void cudaTestFREXP
(
        int max_threads,
        const T *in,
        T *out,
        int *nptr
        )
{
    uint idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < max_threads)
        out[idx] = FREXP(in[idx], &nptr[ idx] );

}

//*****************************************************************
//testLDEXP
//Input:
//max_threads, number of items in in and out arrays
//array of in, of type T
//Output: out array
//*****************************************************************
template<class T>
__global__
void cudaTestLDEXP(
        int max_threads,
        const T *in,
        T *out
        )
{
    uint idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < max_threads)
        out[idx] = LDEXP(in[idx], 10);
}

template<class T>
void testFREXP(
        device_vector<T> &in,
        device_vector<T> &out)
{
    ErrorCheck ec;
    device_vector<int> d_vec_nptr(nx*ny*nz);
    hipEvent_t start, stop;
    float millisecs;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord( start, 0 );
    //stupid laptop with max dim of grid size of 2^15
    const int block_size = 512;
    const int grid_size = nx*ny*nz / block_size;

    cudaTestFREXP<T><<<grid_size, block_size>>>(
        nx*ny*nz,
        raw_pointer_cast(in.data()),
        raw_pointer_cast(out.data()),
        raw_pointer_cast(d_vec_nptr.data())
    ); ec.chk("testFREXP");

    T sum = reduce(
            out.begin(),
            out.end()
        );
    hipStreamSynchronize(0);

    hipEventRecord( stop, 0 );
    hipEventSynchronize(stop);
    hipEventElapsedTime( &millisecs, start, stop );

    cout << "FREXP sum: " << sum << " in time: " << time << endl;

    hipEventDestroy( start );
    hipEventDestroy( stop );

}

template<class T>
void testLDEXP(
        device_vector<T> &in,
        device_vector<T> &out)
{
    ErrorCheck ec;
    hipEvent_t start, stop;
    float millisecs;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord( start, 0 );

    //stupid laptop with max dim of grid size of 2^15
    const int block_size = 512;
    const int grid_size = nx*ny*nz / block_size;
    ec.chk("pre-testLDEXP");
    cudaTestLDEXP<T><<<grid_size, block_size>>>(
        nx*ny*nz,
        raw_pointer_cast(in.data()),
        raw_pointer_cast(out.data())
    ); ec.chk("testLDEXP");
    T sum = reduce(
                out.begin(),
                out.end()
    );
    hipStreamSynchronize(0);

    hipEventRecord( stop, 0 );
    hipEventSynchronize(stop);
    hipEventElapsedTime( &millisecs, start, stop );
    cout << "LDEXP sum: " << sum << " in time: " << millisecs << endl;

}

int main()
{
    device_vector<double> d_vec_in(nx*ny*nz), d_vec_out(nx*ny*nz);
    host_vector<double> h_vec(nx*ny*nz);

    thrust::counting_iterator<uint> index_sequence_begin(0);
    thrust::transform(
                    index_sequence_begin,
                    index_sequence_begin + nx*ny*nz,
                    d_vec_in.begin(),
                    RandGen());

    testFREXP<double>(d_vec_in, d_vec_out);
    testLDEXP<double>(d_vec_in, d_vec_out);

}
