#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define KEPLER 0
#include "ErrorCheck.h"
#include "include/encode.cuh"
#include "include/decode.cuh"

using namespace thrust;
using namespace std;

#define FREXP(x, e) frexp(x, e)
#define LDEXP(x, e) ldexp(x, e)
#define index(x, y, z) ((x) + 4 * ((y) + 4 * (z)))

const int nx = 256;
const int ny = 256;
const int nz = 256;

static const unsigned char
perm[64] = {
  index(0, 0, 0), //  0 : 0

  index(1, 0, 0), //  1 : 1
  index(0, 1, 0), //  2 : 1
  index(0, 0, 1), //  3 : 1

  index(0, 1, 1), //  4 : 2
  index(1, 0, 1), //  5 : 2
  index(1, 1, 0), //  6 : 2

  index(2, 0, 0), //  7 : 2
  index(0, 2, 0), //  8 : 2
  index(0, 0, 2), //  9 : 2

  index(1, 1, 1), // 10 : 3

  index(2, 1, 0), // 11 : 3
  index(2, 0, 1), // 12 : 3
  index(0, 2, 1), // 13 : 3
  index(1, 2, 0), // 14 : 3
  index(1, 0, 2), // 15 : 3
  index(0, 1, 2), // 16 : 3

  index(3, 0, 0), // 17 : 3
  index(0, 3, 0), // 18 : 3
  index(0, 0, 3), // 19 : 3

  index(2, 1, 1), // 20 : 4
  index(1, 2, 1), // 21 : 4
  index(1, 1, 2), // 22 : 4

  index(0, 2, 2), // 23 : 4
  index(2, 0, 2), // 24 : 4
  index(2, 2, 0), // 25 : 4

  index(3, 1, 0), // 26 : 4
  index(3, 0, 1), // 27 : 4
  index(0, 3, 1), // 28 : 4
  index(1, 3, 0), // 29 : 4
  index(1, 0, 3), // 30 : 4
  index(0, 1, 3), // 31 : 4

  index(1, 2, 2), // 32 : 5
  index(2, 1, 2), // 33 : 5
  index(2, 2, 1), // 34 : 5

  index(3, 1, 1), // 35 : 5
  index(1, 3, 1), // 36 : 5
  index(1, 1, 3), // 37 : 5

  index(3, 2, 0), // 38 : 5
  index(3, 0, 2), // 39 : 5
  index(0, 3, 2), // 40 : 5
  index(2, 3, 0), // 41 : 5
  index(2, 0, 3), // 42 : 5
  index(0, 2, 3), // 43 : 5

  index(2, 2, 2), // 44 : 6

  index(3, 2, 1), // 45 : 6
  index(3, 1, 2), // 46 : 6
  index(1, 3, 2), // 47 : 6
  index(2, 3, 1), // 48 : 6
  index(2, 1, 3), // 49 : 6
  index(1, 2, 3), // 50 : 6

  index(0, 3, 3), // 51 : 6
  index(3, 0, 3), // 52 : 6
  index(3, 3, 0), // 53 : 6

  index(3, 2, 2), // 54 : 7
  index(2, 3, 2), // 55 : 7
  index(2, 2, 3), // 56 : 7

  index(1, 3, 3), // 57 : 7
  index(3, 1, 3), // 58 : 7
  index(3, 3, 1), // 59 : 7

  index(2, 3, 3), // 60 : 8
  index(3, 2, 3), // 61 : 8
  index(3, 3, 2), // 62 : 8

  index(3, 3, 3), // 63 : 9
};



void setupConst(const unsigned char *perm)
{
    ErrorCheck ec;
    ec.chk("setupConst start");
    hipMemcpyToSymbol(HIP_SYMBOL(c_perm), perm, sizeof(unsigned char)*64,0); ec.chk("setupConst: lic_dim");
    ec.chk("setupConst finished");


}



//Used to generate rand array in CUDA with Thrust
struct RandGen
{
    RandGen() {}

    __device__ float operator () (const uint idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist(0.0, 0.0001);
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

template<class Int>
void gpuInvXform
(
        device_vector<Int> &q
        )
{
//    ErrorCheck ec;
//    dim3 block_size, grid_size;
//    uint tot_size = 0;

//    tot_size = nx*ny*nz;
//    tot_size /= 64;
//    block_size = dim3(8, 8, 16);
//    grid_size.x = sqrt(tot_size);
//    grid_size.y = sqrt(tot_size);
//    grid_size.z = 1;
//    grid_size.x /= block_size.x; grid_size.y /= block_size.y;

//    cout << grid_size.x << " " << grid_size.y << " " << grid_size.z << endl;
//    cudaInvXFormYX<Int> << <grid_size, block_size >> >
//        (
//        raw_pointer_cast(q.data())
//        );
//    hipStreamSynchronize(0);
//    ec.chk("cudaInvXFormYX");

//    tot_size = nx*ny*nz;

//    tot_size /= 16;
//    block_size = dim3(8, 8, 4);
//    grid_size.x = sqrt(tot_size);
//    grid_size.y = sqrt(tot_size);
//    grid_size.z = 1;
//    grid_size.x /= block_size.x; grid_size.y /= block_size.y;

//    cout << grid_size.x << " " << grid_size.y << " " << grid_size.z << endl;
//    cudaInvXFormXZ<Int> << <grid_size, block_size >> >
//        (
//        raw_pointer_cast(q.data())
//        );
//    hipStreamSynchronize(0);
//    ec.chk("cudaInvXFormXZ");

//    block_size = dim3(8, 8, 8);
//    grid_size = dim3(nx,ny,nz);
//    grid_size.x /= block_size.x; grid_size.y /= block_size.y; grid_size.z /= block_size.z;
//    grid_size.z /= 4;

//    cout << grid_size.x << " " << grid_size.y << " " << grid_size.z << endl;
//    cudaInvXFormZY<Int> << <grid_size, block_size >> >
//        (
//        raw_pointer_cast(q.data())
//        );
//    hipStreamSynchronize(0);
//    ec.chk("cudaInvXFormZY");


    ErrorCheck ec;
    dim3 emax_size(nx / 4, ny / 4, nz / 4);
    dim3 block_size, grid_size;

    block_size = dim3(8,8,8);
    grid_size = emax_size;
    grid_size.x /= block_size.x; grid_size.y /= block_size.y;  grid_size.z /= block_size.z;

    cudaInvXForm<Int><<<grid_size, block_size>>>
        (
            raw_pointer_cast(q.data())
        );
    hipStreamSynchronize(0);
    ec.chk("cudaInvXForm");

}


template<class Int>
void gpuTestinv_xform
(
        host_vector<Int> &h_q
        )
{
	ErrorCheck ec;
	device_vector<Int> q_out;
	q_out.resize(nx*ny*nz);
	q_out = h_q;
	dim3 emax_size(nx / 4, ny / 4, nz / 4);

	dim3 block_size(8, 8, 8);
	dim3 grid_size = emax_size;
	grid_size.x /= block_size.x; grid_size.y /= block_size.y;  grid_size.z /= block_size.z;


//	cudaInvXForm<Int> << <block_size, grid_size >> >
//		(
//		raw_pointer_cast(q_out.data())
//		);
//	hipStreamSynchronize(0);
//	ec.chk("cudaInvXForm");

    float millisecs;
    hipEvent_t start_decode, stop_decode;
    hipEventCreate(&start_decode);
    hipEventCreate(&stop_decode);
    hipEventRecord(start_decode, 0);

    gpuInvXform(q_out);
    hipEventRecord(stop_decode, 0);
    hipEventSynchronize(stop_decode);
    hipEventElapsedTime(&millisecs, start_decode, stop_decode);

    cout << "inv_xform GPU in time (in ms): " << millisecs << endl;

	host_vector<Int> h_qout;

	h_qout = q_out;
	std::vector<Int> iblock;
	iblock.resize(h_q.size());
	thrust::copy(h_q.begin(), h_q.end(), iblock.begin());
	for (int i = 0; i < nx*ny*nz / 64; i++){
        inv_xform(&iblock[0] + i * 64);
//        inv_xform_yx(&iblock[0] + i * 64);
//        inv_xform_xz(&iblock[0] + i * 64);
//        inv_xform_zy(&iblock[0] + i * 64);
    }
	int i = 0;
	for (i = 0; i < nx*ny*nz; i++){
        if(iblock[i] != h_qout[i]){
            cout << i << " " << iblock[i] << " " << h_qout[i] << endl;
            exit(-1);
        }
	}
}

typedef long long Int;

int main()
{

    host_vector<Int> h_q;
    h_q.resize(nx*ny*nz);
    for (int i=0; i<h_q.size(); i++){
        h_q[i] = i;
    }



    setupConst(perm);
    gpuTestinv_xform<long long>(h_q);



}
