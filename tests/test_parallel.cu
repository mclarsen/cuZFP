#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <omp.h>

#define KEPLER 0
#include "ErrorCheck.h"
#include "include/encode.cuh"
#include "include/decode.cuh"

using namespace thrust;
using namespace std;
using namespace cuZFP;

#define index(x, y, z) ((x) + 4 * ((y) + 4 * (z)))

const size_t nx = 128;
const size_t ny = 128;
const size_t nz = 128;

uint minbits = 1024;
uint maxbits = 1024;
uint MAXPREC = 64;
int MINEXP = -1074;
const double rate = 64;
size_t  blksize = 0;
unsigned long long group_count = 0x46acca631ull;
uint size = 64;
int EBITS = 11;                     /* number of exponent bits */


static const unsigned char
perm[64] = {
	index(0, 0, 0), //  0 : 0

	index(1, 0, 0), //  1 : 1
	index(0, 1, 0), //  2 : 1
	index(0, 0, 1), //  3 : 1

	index(0, 1, 1), //  4 : 2
	index(1, 0, 1), //  5 : 2
	index(1, 1, 0), //  6 : 2

	index(2, 0, 0), //  7 : 2
	index(0, 2, 0), //  8 : 2
	index(0, 0, 2), //  9 : 2

	index(1, 1, 1), // 10 : 3

	index(2, 1, 0), // 11 : 3
	index(2, 0, 1), // 12 : 3
	index(0, 2, 1), // 13 : 3
	index(1, 2, 0), // 14 : 3
	index(1, 0, 2), // 15 : 3
	index(0, 1, 2), // 16 : 3

	index(3, 0, 0), // 17 : 3
	index(0, 3, 0), // 18 : 3
	index(0, 0, 3), // 19 : 3

	index(2, 1, 1), // 20 : 4
	index(1, 2, 1), // 21 : 4
	index(1, 1, 2), // 22 : 4

	index(0, 2, 2), // 23 : 4
	index(2, 0, 2), // 24 : 4
	index(2, 2, 0), // 25 : 4

	index(3, 1, 0), // 26 : 4
	index(3, 0, 1), // 27 : 4
	index(0, 3, 1), // 28 : 4
	index(1, 3, 0), // 29 : 4
	index(1, 0, 3), // 30 : 4
	index(0, 1, 3), // 31 : 4

	index(1, 2, 2), // 32 : 5
	index(2, 1, 2), // 33 : 5
	index(2, 2, 1), // 34 : 5

	index(3, 1, 1), // 35 : 5
	index(1, 3, 1), // 36 : 5
	index(1, 1, 3), // 37 : 5

	index(3, 2, 0), // 38 : 5
	index(3, 0, 2), // 39 : 5
	index(0, 3, 2), // 40 : 5
	index(2, 3, 0), // 41 : 5
	index(2, 0, 3), // 42 : 5
	index(0, 2, 3), // 43 : 5

	index(2, 2, 2), // 44 : 6

	index(3, 2, 1), // 45 : 6
	index(3, 1, 2), // 46 : 6
	index(1, 3, 2), // 47 : 6
	index(2, 3, 1), // 48 : 6
	index(2, 1, 3), // 49 : 6
	index(1, 2, 3), // 50 : 6

	index(0, 3, 3), // 51 : 6
	index(3, 0, 3), // 52 : 6
	index(3, 3, 0), // 53 : 6

	index(3, 2, 2), // 54 : 7
	index(2, 3, 2), // 55 : 7
	index(2, 2, 3), // 56 : 7

	index(1, 3, 3), // 57 : 7
	index(3, 1, 3), // 58 : 7
	index(3, 3, 1), // 59 : 7

	index(2, 3, 3), // 60 : 8
	index(3, 2, 3), // 61 : 8
	index(3, 3, 2), // 62 : 8

	index(3, 3, 3), // 63 : 9
};


static size_t block_size(double rate) { return (lrint(64 * rate) + CHAR_BIT - 1) / CHAR_BIT; }


template<class Scalar>
void setupConst(const unsigned char *perm,
	uint maxprec_,
	int minexp_,
	int ebits_)
{
	ErrorCheck ec;
	ec.chk("setupConst start");
	hipMemcpyToSymbol(HIP_SYMBOL(c_perm), perm, sizeof(unsigned char) * 64, 0); ec.chk("setupConst: c_perm");

	const uint sizeof_scalar = sizeof(Scalar);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sizeof_scalar), &sizeof_scalar, sizeof(uint)); ec.chk("setupConst: c_sizeof_scalar");

	hipMemcpyToSymbol(HIP_SYMBOL(c_maxprec), &maxprec_, sizeof(uint)); ec.chk("setupConst: c_maxprec");
	hipMemcpyToSymbol(HIP_SYMBOL(c_minexp), &minexp_, sizeof(int)); ec.chk("setupConst: c_minexp");
	hipMemcpyToSymbol(HIP_SYMBOL(c_ebits), &ebits_, sizeof(int)); ec.chk("setupConst: c_ebits");

	ec.chk("setupConst finished");



}

template<class Int, class UInt, class Scalar, uint bsize>
void cpuEFPDI2UTransform
(
const dim3 &emax_size,
const dim3 &blockDim,
const dim3 &gridDim,
const Scalar *data,
UInt *p,
Bit<bsize> *stream

)
{
	uint3 blockIdx;

	for (blockIdx.z = 0; blockIdx.z < gridDim.z; blockIdx.z++){
		for (blockIdx.y = 0; blockIdx.y < gridDim.y; blockIdx.y++){
			for (blockIdx.x = 0; blockIdx.x < gridDim.x; blockIdx.x++){
				uint3 threadIdx;
				//extern __shared__ long long sh_q[];
				long long *sh_q = new long long[64*64];
				for (threadIdx.z = 0; threadIdx.z < blockDim.z; threadIdx.z++){
					for (threadIdx.y = 0; threadIdx.y < blockDim.y; threadIdx.y++){
						for (threadIdx.x = 0; threadIdx.x < blockDim.x; threadIdx.x++){
							int mx = threadIdx.x + blockDim.x*blockIdx.x;
							int my = threadIdx.y + blockDim.y*blockIdx.y;
							int mz = threadIdx.z + blockDim.z*blockIdx.z;
							int eidx = mz*gridDim.x*blockDim.x*gridDim.y*blockDim.y + my*gridDim.x*blockDim.x + mx;

							mx *= 4; my *= 4; mz *= 4;
							//int idx = z*gridDim.x*gridDim.y*blockDim.x*blockDim.y*16 + y*gridDim.x*blockDim.x*4+ x;
							int emax = max_exp_block(data, mx, my, mz, 1, gridDim.x*blockDim.x * 4, gridDim.x*blockDim.x * 4 * gridDim.y*blockDim.y * 4);

							stream[eidx].emax = emax;
							//	uint sz = gridDim.x*blockDim.x * 4 * gridDim.y*blockDim.y * 4;
							//	uint sy = gridDim.x*blockDim.x * 4;
							//	uint sx = 1;
							fixed_point_block(sh_q + (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 16) * 64, data, emax, mx, my, mz, 1, gridDim.x*blockDim.x * 4, gridDim.x*blockDim.x * 4 * gridDim.y*blockDim.y * 4);
							fwd_xform(sh_q + (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 16) * 64);


							//fwd_order
							for (int i = 0; i < 64; i++){
								uint idx = eidx * 64 + i;
								p[idx] = int2uint<Int, UInt>(sh_q[(threadIdx.x + threadIdx.y * 4 + threadIdx.z * 16) * 64 + perm[i]]);
							}
						}
					}
				}
				delete[]sh_q;


			}
		}
	}
}


/* reorder unsigned coefficients and convert to signed integer */
template<class Int, class UInt>
__host__
static void
inv_order(const UInt* ublock, Int* iblock, const unsigned char* perm, uint n)
{
	do
		iblock[*perm++] = uint2int<UInt>(*ublock++);
	while (--n);
}


//Used to generate rand array in CUDA with Thrust
struct RandGen
{
	RandGen() {}

	__device__ float operator () (const uint idx)
	{
		thrust::default_random_engine randEng;
		thrust::uniform_real_distribution<float> uniDist(0.0, 0.0001);
		randEng.discard(idx);
		return uniDist(randEng);
	}
};
template<class Int, class UInt>
void reorder
(
const Int *q,
UInt *buffer
)
{
	for (uint i = 0; i < 64; i++)
		buffer[i] = int2uint<Int, UInt>(q[perm[i]]);
}


template<class UInt, uint bsize>
void cpuEncodeUInt
(
const unsigned long long count,
uint size,
const UInt* data,
const unsigned char *g_cnt,
Bit<bsize> *stream
)
{

	//extern __shared__ unsigned char smem[];
	//__shared__ unsigned char *sh_g, *sh_sbits;
	//__shared__ Bitter *sh_bitters;

	//sh_g = &smem[0];
	//sh_sbits = &smem[64];
	//sh_bitters = (Bitter*)&smem[64 + 64];

	//uint tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z *blockDim.x*blockDim.y;

	//uint bidx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x)*blockDim.x*blockDim.y*blockDim.z;

	uint3 blockIdx, gridDim, blockDim;
	gridDim.x = nx / 4;
	gridDim.y = ny / 4;
	gridDim.z = nz / 4;
	
	blockDim.x = blockDim.y = blockDim.z = 4;

	for (blockIdx.z = 0; blockIdx.z < gridDim.z; blockIdx.z++){
		for (blockIdx.y = 0; blockIdx.y < gridDim.y; blockIdx.y++){
			for (blockIdx.x = 0; blockIdx.x <gridDim.x; blockIdx.x++){
				uint bidx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x)*blockDim.x*blockDim.y*blockDim.z;

				unsigned long long x[64];
				Bitter bitter[64];
				unsigned char sbit[64];
				for (int i = 0; i < 64; i++){
					bitter[i] = make_bitter(0, 0);
					sbit[i] = 0;
				}
				uint s_emax_bits[1];
				s_emax_bits[0] = 1;
				//maxprec, minexp, EBITS
				//	uint k = threadIdx.x + blockDim.x * blockIdx.x;
				int emax = stream[bidx / 64].emax;
				int maxprec = precision(emax, MAXPREC, MINEXP);
				int ebits = EBITS + 1;
				const uint kmin = intprec > maxprec ? intprec - maxprec : 0;

				uint e = maxprec ? emax + ebias : 0;
				//printf("%d %d %d %d\n", emax, maxprec, ebits, e);
				if (e){
					//write_bitters(bitter[0], make_bitter(2 * e + 1, 0), ebits, sbit[0]);
					stream[bidx / 64].begin[0] = 2 * e + 1;
					s_emax_bits[0] = ebits;
				}
//				const uint kmin = intprec > MAXPREC ? intprec - MAXPREC : 0;

				unsigned long long y[64];
				for (int tid = 0; tid < 64; tid++){
					/* extract bit plane k to x[k] */
					y[tid] = 0;
					for (uint i = 0; i < size; i++)
						y[tid] += ((data[bidx + i] >> tid) & (unsigned long long)1) << i;
					x[tid] = y[tid];
				}

				char sh_g[64], sh_sbits[64];
				Bitter sh_bitters[64];

				/* count number of positive group tests g[k] among 3*d in d dimensions */
				for (int tid = 0; tid < 64; tid++){
					sh_g[tid] = 0;
					for (unsigned long long c = count; y[tid]; y[tid] >>= c & 0xfu, c >>= 4)
						sh_g[tid]++;
				}


				unsigned char cur = sh_g[intprec - 1];

				for (int i = intprec - 1; i-- > kmin;) {
					if (cur < sh_g[i])
						cur = sh_g[i];
					else if (cur > sh_g[i])
						sh_g[i] = cur;
				}

				for (int tid = 0; tid < 64; tid++){
					unsigned char g = sh_g[tid];
					unsigned char h = sh_g[min(tid + 1, intprec - 1)];


					encodeBitplane(count, x[tid], g, h, g_cnt, bitter[tid], sbit[tid]);
					sh_bitters[63 - tid] = bitter[tid];
					sh_sbits[63 - tid] = sbit[tid];
				}


				uint tot_sbits = s_emax_bits[0];
				uint offset = 0;
				for (int i = 0; i < intprec; i++){
					if (sh_sbits[i] <= 64){
						write_outx(sh_bitters, stream[bidx / 64].begin, tot_sbits, offset, i, sh_sbits[i]);
					}
					else{
						write_outx(sh_bitters, stream[bidx / 64].begin, tot_sbits, offset, i, 64);
						write_outy(sh_bitters, stream[bidx / 64].begin, tot_sbits, offset, i, sh_sbits[i] - 64);
					}
				}
			}
		}
	}
}


template<class Int, class UInt, class Scalar, uint bsize>
void cpuEncode
(
dim3 gridDim, 
dim3 blockDim,
const unsigned long long count,
uint size,
const Scalar* data,
const unsigned char *g_cnt,
Bit<bsize> *stream
)
{

	//extern __shared__ unsigned char smem[];
	//__shared__ unsigned char *sh_g, *sh_sbits;
	//__shared__ Bitter *sh_bitters;

	//sh_g = &smem[0];
	//sh_sbits = &smem[64];
	//sh_bitters = (Bitter*)&smem[64 + 64];

	//uint tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z *blockDim.x*blockDim.y;

	//uint bidx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x)*blockDim.x*blockDim.y*blockDim.z;

	dim3 blockIdx;
	
	for (blockIdx.z = 0; blockIdx.z < gridDim.z; blockIdx.z++){
		for (blockIdx.y = 0; blockIdx.y < gridDim.y; blockIdx.y++){
			for (blockIdx.x = 0; blockIdx.x <gridDim.x; blockIdx.x++){


				Int sh_q[64];
				UInt sh_p[64];
				uint mx = blockIdx.x, my = blockIdx.y, mz = blockIdx.z;
				mx *= 4; my *= 4; mz *= 4;
				int emax = max_exp_block(data, mx, my, mz, 1, blockDim.x * gridDim.x, gridDim.x * gridDim.y * blockDim.x * blockDim.y);

				//	uint sz = gridDim.x*blockDim.x * 4 * gridDim.y*blockDim.y * 4;
				//	uint sy = gridDim.x*blockDim.x * 4;
				//	uint sx = 1;
				fixed_point_block(sh_q, data, emax, mx, my, mz, 1, blockDim.x * gridDim.x, gridDim.x  * gridDim.y * blockDim.x * blockDim.y);
				fwd_xform(sh_q);


				//fwd_order
				for (int i = 0; i < 64; i++){
					sh_p[i] = int2uint<Int, UInt>(sh_q[perm[i]]);
				}


				uint bidx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x);

				unsigned long long x[64];
				Bitter bitter[64];
				unsigned char sbit[64];
				for (int i = 0; i < 64; i++){
					bitter[i] = make_bitter(0, 0);
					sbit[i] = 0;
				}
				uint s_emax_bits[1];
				s_emax_bits[0] = 1;
				//maxprec, minexp, EBITS
				//	uint k = threadIdx.x + blockDim.x * blockIdx.x;
				int maxprec = precision(emax, MAXPREC, MINEXP);
				int ebits = EBITS + 1;
				const uint kmin = intprec > maxprec ? intprec - maxprec : 0;

				uint e = maxprec ? emax + ebias : 0;
				//printf("%d %d %d %d\n", emax, maxprec, ebits, e);
				if (e){
					//write_bitters(bitter[0], make_bitter(2 * e + 1, 0), ebits, sbit[0]);
					stream[bidx].begin[0] = 2 * e + 1;
					s_emax_bits[0] = ebits;
				}
				//				const uint kmin = intprec > MAXPREC ? intprec - MAXPREC : 0;

				unsigned long long y[64];
				for (int tid = 0; tid < 64; tid++){
					/* extract bit plane k to x[k] */
					y[tid] = 0;
					for (uint i = 0; i < size; i++)
						y[tid] += ((sh_p[i] >> tid) & (unsigned long long)1) << i;
					x[tid] = y[tid];
				}

				char sh_g[64], sh_sbits[64];
				Bitter sh_bitters[64];

				/* count number of positive group tests g[k] among 3*d in d dimensions */
				for (int tid = 0; tid < 64; tid++){
					sh_g[tid] = 0;
					for (unsigned long long c = count; y[tid]; y[tid] >>= c & 0xfu, c >>= 4)
						sh_g[tid]++;
				}


				unsigned char cur = sh_g[intprec - 1];

				for (int i = intprec - 1; i-- > kmin;) {
					if (cur < sh_g[i])
						cur = sh_g[i];
					else if (cur > sh_g[i])
						sh_g[i] = cur;
				}

				for (int tid = 0; tid < 64; tid++){
					unsigned char g = sh_g[tid];
					unsigned char h = sh_g[min(tid + 1, intprec - 1)];


					encodeBitplane(count, x[tid], g, h, g_cnt, bitter[tid], sbit[tid]);
					sh_bitters[63 - tid] = bitter[tid];
					sh_sbits[63 - tid] = sbit[tid];
				}


				uint tot_sbits = s_emax_bits[0];
				uint offset = 0;
				for (int i = 0; i < intprec; i++){
					if (sh_sbits[i] <= 64){
						write_outx(sh_bitters, stream[bidx].begin, tot_sbits, offset, i, sh_sbits[i]);
					}
					else{
						write_outx(sh_bitters, stream[bidx].begin, tot_sbits, offset, i, 64);
						write_outy(sh_bitters, stream[bidx].begin, tot_sbits, offset, i, sh_sbits[i] - 64);
					}
				}
			}
		}
	}
}
template<class Int, class UInt, uint bsize, uint num_sidx>
void cpuDecodeInvOrder
(
size_t *sidx,
Bit<bsize> *stream,

Int *data,

const unsigned long long orig_count

)
{
	//uint tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z *blockDim.x*blockDim.y;
	//uint idx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x);
	//uint bdim = blockDim.x*blockDim.y*blockDim.z;
	//uint bidx = idx*bdim;

	uint3 blockIdx, gridDim, blockDim;
	gridDim.x = nx / 4;
	gridDim.y = ny / 4;
	gridDim.z = nz / 4;

	blockDim.x = blockDim.y = blockDim.z = 4;

	for (blockIdx.z = 0; blockIdx.z < gridDim.z; blockIdx.z++){
		for (blockIdx.y = 0; blockIdx.y < gridDim.y; blockIdx.y++){
			for (blockIdx.x = 0; blockIdx.x < gridDim.x; blockIdx.x++){
				uint idx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x);
				uint bdim = blockDim.x*blockDim.y*blockDim.z;
				uint bidx = idx*bdim;

				size_t s_sidx[64];// = (size_t*)&smem[0];
				//if (tid < num_sidx)
				for (int tid = 0; tid < num_sidx; tid++){

					s_sidx[tid] = sidx[tid];
				}

				uint s_idx_n[64];// = (uint*)&smem[s_sidx[0]];
				uint s_idx_g[64];// = (uint*)&smem[s_sidx[1]];
				unsigned long long s_bit_cnt[64];// = (unsigned long long*)&smem[s_sidx[2]];
				uint s_bit_rmn_bits[64];// = (uint*)&smem[s_sidx[3]];
				char s_bit_offset[64];// = (char*)&smem[s_sidx[4]];
				uint s_bit_bits[64];// = (uint*)&smem[s_sidx[5]];
				Word s_bit_buffer[64];// = (Word*)&smem[s_sidx[6]];
				UInt s_data[64];// = (UInt*)&smem[s_sidx[7]];
				uint s_kmin[1];


				stream[idx].read_bit();
				uint ebits = EBITS + 1;
				int emax = stream[idx].read_bits(ebits - 1) - ebias;
				int maxprec = precision(emax, MAXPREC, MINEXP);
				s_kmin[0] = intprec > maxprec ? intprec - maxprec : 0;

				
				for (int tid = 0; tid < 64; tid++){
					s_idx_g[tid] = 0;
					s_data[tid] = 0;
				}

				insert_bit<bsize>(
					stream[idx],
					s_idx_g,
					s_idx_n,
					s_bit_bits,
					s_bit_offset,
					s_bit_buffer,
					s_bit_cnt,
					s_bit_rmn_bits,
					maxbits - ebits, intprec, s_kmin[0], orig_count);

				for (int tid = 0; tid < 64; tid++){

					for (uint k = s_kmin[0]; k < intprec; k++){
						decodeBitstream<UInt, bsize>(
							stream[idx],
							s_idx_g[k],
							s_idx_n[k],
							s_bit_cnt[k],
							s_bit_rmn_bits[k],
							s_bit_bits[k],
							s_bit_offset[k],
							s_bit_buffer[k],
							s_data[tid],
							tid, k);
					}

					data[perm[tid] + bidx] = uint2int<Int, UInt>(s_data[tid]);
				}
			}
		}
	}
}

template<class Int, class UInt, class Scalar, uint bsize, uint num_sidx>
void cpuDecode
(
dim3 gridDim,
dim3 blockDim,
size_t *sidx,
Bit<bsize> *stream,

Scalar *out,
const unsigned long long orig_count

)
{
	//uint tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z *blockDim.x*blockDim.y;
	//uint idx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x);
	//uint bdim = blockDim.x*blockDim.y*blockDim.z;
	//uint bidx = idx*bdim;

	dim3 blockIdx;

	for (blockIdx.z = 0; blockIdx.z < gridDim.z; blockIdx.z++){
		for (blockIdx.y = 0; blockIdx.y < gridDim.y; blockIdx.y++){
			for (blockIdx.x = 0; blockIdx.x < gridDim.x; blockIdx.x++){
				uint idx = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x);
				uint bdim = blockDim.x*blockDim.y*blockDim.z;
				uint bidx = idx*bdim;

				size_t s_sidx[64];// = (size_t*)&smem[0];
				//if (tid < num_sidx)
				for (int tid = 0; tid < num_sidx; tid++){

					s_sidx[tid] = sidx[tid];
				}

				uint s_idx_n[64];// = (uint*)&smem[s_sidx[0]];
				uint s_idx_g[64];// = (uint*)&smem[s_sidx[1]];
				unsigned long long s_bit_cnt[64];// = (unsigned long long*)&smem[s_sidx[2]];
				uint s_bit_rmn_bits[64];// = (uint*)&smem[s_sidx[3]];
				char s_bit_offset[64];// = (char*)&smem[s_sidx[4]];
				uint s_bit_bits[64];// = (uint*)&smem[s_sidx[5]];
				Word s_bit_buffer[64];// = (Word*)&smem[s_sidx[6]];
				UInt s_data[64];// = (UInt*)&smem[s_sidx[7]];
				Int s_q[64];
				uint s_kmin[1];
				int s_emax[1];


				stream[idx].read_bit();
				uint ebits = EBITS + 1;
				s_emax[0] = stream[idx].read_bits(ebits - 1) - ebias;
				int maxprec = precision(s_emax[0], MAXPREC, MINEXP);
				s_kmin[0] = intprec > maxprec ? intprec - maxprec : 0;


				for (int tid = 0; tid < 64; tid++){
					s_idx_g[tid] = 0;
					s_data[tid] = 0;
				}

				insert_bit<bsize>(
					stream[idx],
					s_idx_g,
					s_idx_n,
					s_bit_bits,
					s_bit_offset,
					s_bit_buffer,
					s_bit_cnt,
					s_bit_rmn_bits,
					maxbits - ebits, intprec, s_kmin[0], orig_count);

				for (int tid = 0; tid < 64; tid++){

					for (uint k = s_kmin[0]; k < intprec; k++){
						decodeBitstream<UInt, bsize>(
							stream[idx],
							s_idx_g[k],
							s_idx_n[k],
							s_bit_cnt[k],
							s_bit_rmn_bits[k],
							s_bit_bits[k],
							s_bit_offset[k],
							s_bit_buffer[k],
							s_data[tid],
							tid, k);
					}

					s_q[perm[tid]] = uint2int<Int, UInt>(s_data[tid]);




				}

				uint mx = blockIdx.x, my = blockIdx.y, mz = blockIdx.z;
				mx *= 4; my *= 4; mz *= 4;

				inv_xform(s_q);
				inv_cast<Int, Scalar>(s_q, out, s_emax[0], mx, my, mz, 1, gridDim.x*blockDim.x, gridDim.x*blockDim.x * gridDim.y*blockDim.y);

			}
		}
	}
}
template<class Int, class UInt, class Scalar, uint bsize>
void gpuValidate
(
host_vector<Scalar> &h_p,
device_vector<Int> &q,
device_vector<Scalar> &data
)
{
	host_vector<Int> h_q;

	h_q = q;

	int i = 0;
	for (int z = 0; z < nz; z += 4){
		for (int y = 0; y < ny; y += 4){
			for (int x = 0; x < nx; x += 4){
				int idx = z*nx*ny + y*nx + x;
				host_vector<Int> q2(64);
				host_vector<UInt> buf(64);
				Bit<bsize> loc_stream;
				int emax2 = max_exp_block<Scalar>(raw_pointer_cast(h_p.data()), x, y, z, 1, nx, nx*ny);
				fixed_point_block(raw_pointer_cast(q2.data()), raw_pointer_cast(h_p.data()), emax2, x, y, z, 1, nx, nx*ny);
				fwd_xform(raw_pointer_cast(q2.data()));
				reorder<Int, UInt>(raw_pointer_cast(q2.data()), raw_pointer_cast(buf.data()));
				encode_ints<UInt>(loc_stream, raw_pointer_cast(buf.data()), minbits, maxbits, precision(emax2, maxprec, minexp), group_count, size);

				loc_stream.rewind();
				UInt dec[64];

				decode_ints<UInt, bsize>(loc_stream, dec, minbits, maxbits, precision(emax2, maxprec, minexp), group_count, size);


				Int iblock[64];
				inv_order(dec, iblock, perm, 64);
				inv_xform(iblock);

				for (int j = 0; j < 64; j++){
					assert(h_q[i * 64 + j] == iblock[j]);
				}

				Scalar fblock[64];
				inv_cast(iblock, fblock, emax2, 0, 0, 0, 1, 4, 16);

				int fidx = 0;
				for (int k = z; k < z + 4; k++){
					for (int j = y; j < y + 4; j++){
						for (int i = x; i < x + 4; i++, fidx++){
							if (h_p[k*nz*ny + j*ny + i] != fblock[fidx]){
								cout << "inv_cast failed: " << k << " " << j << " " << i << " " << fidx << " " << h_p[k*nz*ny + j*ny + i] << " " << fblock[fidx] << endl;
								exit(-1);
							}

						}
					}
				}
				i++;

			}
		}
	}
}

template<class Int, class UInt, class Scalar, uint bsize>
void gpuTestBitStream
(
host_vector<Scalar> &h_data
)
{
	host_vector<int> h_emax;
	host_vector<UInt> h_p;
	host_vector<Int> h_q;
	host_vector<UInt> h_buf(nx*ny*nz);
	host_vector<Bit<bsize> > h_bits;
	device_vector<unsigned char> d_g_cnt;

  device_vector<Scalar> data;
  data = h_data;


	dim3 emax_size(nx / 4, ny / 4, nz / 4);

	dim3 block_size(8, 8, 8);
	dim3 grid_size = emax_size;
	grid_size.x /= block_size.x; grid_size.y /= block_size.y;  grid_size.z /= block_size.z;

	//const uint kmin = intprec > maxprec ? intprec - maxprec : 0;

	ErrorCheck ec;

	hipEvent_t start, stop;
	float millisecs;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);



	device_vector<Bit<bsize> > stream(emax_size.x * emax_size.y * emax_size.z);
	host_vector<Bit<bsize> > cpu_stream;

	block_size = dim3(4, 4, 4);
	grid_size = dim3(nx, ny, nz);
	grid_size.x /= block_size.x; grid_size.y /= block_size.y;  grid_size.z /= block_size.z;
	
	unsigned long long count = group_count;
	host_vector<unsigned char> g_cnt(10);
	uint sum = 0;
	g_cnt[0] = 0;
	for (int i = 1; i < 10; i++){
		sum += count & 0xf;
		g_cnt[i] = sum;
		count >>= 4;
	}
	d_g_cnt = g_cnt;
#if 0
	cpu_stream = stream;
	cpuEncode<Int, UInt, Scalar, bsize>(
		grid_size,
		block_size,
		group_count, size,
		thrust::raw_pointer_cast(h_data.data()),
		thrust::raw_pointer_cast(g_cnt.data()),
		thrust::raw_pointer_cast(cpu_stream.data()));
	stream = cpu_stream;
#else
	cudaEncode<Int, UInt,Scalar, bsize> << <grid_size, block_size, (2 * sizeof(unsigned char) + sizeof(Bitter) + sizeof(UInt) + sizeof(Int)) * 64 + 4 >> >
		(
		group_count, size,
		thrust::raw_pointer_cast(data.data()),
		thrust::raw_pointer_cast(d_g_cnt.data()),
		thrust::raw_pointer_cast(stream.data())
		);
	 hipStreamSynchronize(0);
	ec.chk("cudaEncode");
	cpu_stream = stream;
#endif
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&millisecs, start, stop);
	ec.chk("cudaencode");

	cout << "encode GPU in time: " << millisecs << endl;

  hipMemset(thrust::raw_pointer_cast(data.data()), 0, sizeof(Scalar)*data.size());

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	block_size = dim3(8, 8, 8);
	grid_size = emax_size;
	grid_size.x /= block_size.x; grid_size.y /= block_size.y; grid_size.z /= block_size.z;
	cudaRewind<bsize> << < grid_size, block_size >> >
		(
		raw_pointer_cast(stream.data())
		);
	ec.chk("cudaRewind");

#if 1
	block_size = dim3(4, 4, 4);
	grid_size = dim3(nx, ny, nz);
	grid_size.x /= block_size.x; grid_size.y /= block_size.y; grid_size.z /= block_size.z;
	size_t blcksize = block_size.x *block_size.y * block_size.z;
#else
	host_vector<size_t> cpu_sidx = d_sidx;
	host_vector<Int> cpu_q = q;
	cpuDecodeInvOrder < Int, UInt, bsize, 9 >
		(
		raw_pointer_cast(cpu_sidx.data()),
			raw_pointer_cast(cpu_stream.data()),
			raw_pointer_cast(cpu_q.data()),
			group_count);
	stream = cpu_stream;
	q = cpu_q;
#endif

#if 1
	size_t s_idx[12] = { sizeof(size_t) * 12, blcksize * sizeof(uint), blcksize * sizeof(uint), +blcksize * sizeof(unsigned long long), blcksize * sizeof(uint), blcksize * sizeof(char), blcksize * sizeof(uint), blcksize * sizeof(Word), blcksize * sizeof(UInt), blcksize * sizeof(Int), sizeof(uint), sizeof(int) };
	thrust::inclusive_scan(s_idx, s_idx + 11, s_idx);
	const size_t shmem_size = thrust::reduce(s_idx, s_idx + 11);
	device_vector<size_t> d_sidx(s_idx, s_idx + 11);

	cudaDecode<Int, UInt, Scalar, bsize, 11> << < grid_size, block_size, 64 * (4 + 4 + 8 + 4 + 1 + 4 + 8 + 8 + 8) + 4 + 4 >> >

		(
		raw_pointer_cast(d_sidx.data()),
		raw_pointer_cast(stream.data()),
		raw_pointer_cast(data.data()),
		maxbits,
		intprec,
		group_count);
	hipStreamSynchronize(0);
#else
	cpuDecode < Int, UInt, Scalar, bsize, 9 >
		(grid_size, block_size,
		raw_pointer_cast(cpu_sidx.data()),
		raw_pointer_cast(cpu_stream.data()),
		raw_pointer_cast(h_data.data()),
		group_count);
	data = h_data;
#endif
  ec.chk("cudaDecodeInvOrder");
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&millisecs, start, stop);
	ec.chk("cudadecode");
	cout << "decode parallel GPU in time: " << millisecs << endl;

	host_vector<Scalar> h_out = data;
	for (int i = 0; i < h_data.size(); i++){
		if (h_data[i] != h_out[i]){
			cout << i << " " << h_data[i] << " " << h_out[i] << endl;
			exit(-1);
		}
	}
	//gpuValidate<Int, UInt, Scalar, bsize>(h_data, q, data);

}

int main()
{

  host_vector<double> h_vec_in(nx*ny*nz);
  for (int z=0; z<nz; z++){
    for (int y=0; y<ny; y++){
      for (int x=0; x<nx; x++){
        if (x == 0)
          h_vec_in[z*nx*ny + y*nx + x] = 10;
        else if(x == nx - 1)
          h_vec_in[z*nx*ny + y*nx + x] = 0;
        else
          h_vec_in[z*nx*ny + y*nx + x] = 5;

      }
    }
  }
  device_vector<double> d_vec_in;
  d_vec_in = h_vec_in;

//	thrust::counting_iterator<uint> index_sequence_begin(0);
//	thrust::transform(
//		index_sequence_begin,
//		index_sequence_begin + nx*ny*nz,
//		d_vec_in.begin(),
//		RandGen());

	h_vec_in = d_vec_in;
	d_vec_in.clear();
	d_vec_in.shrink_to_fit();
	//    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	setupConst<double>(perm, MAXPREC, MINEXP, EBITS);
	cout << "Begin gpuTestBitStream" << endl;
	gpuTestBitStream<long long int, unsigned long long int, double, 64>(h_vec_in);
	cout << "Finish gpuTestBitStream" << endl;
	//    cout << "Begin cpuTestBitStream" << endl;
	//    cpuTestBitStream<long long, unsigned long long, double, 64>(h_vec_in);
	//    cout << "End cpuTestBitStream" << endl;

	//cout << "Begin gpuTestHarnessSingle" << endl;
	//gpuTestharnessSingle<long long, unsigned long long, double, 64>(h_vec_in, d_vec_out, d_vec_in, 0,0,0);
	//cout << "Begin gpuTestHarnessMulti" << endl;
	//gpuTestharnessMulti<long long, unsigned long long, double, 64>(d_vec_in);
}
