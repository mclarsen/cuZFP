#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>

#define KEPLER 0
#include "ErrorCheck.h"
#include "fixed_point.cuh"

using namespace thrust;
using namespace std;

#define FREXP(x, e) frexp(x, e)
#define LDEXP(x, e) ldexp(x, e)

const int nx = 256;
const int ny = 256;
const int nz = 256;


//Used to generate rand array in CUDA with Thrust
struct RandGen
{
    RandGen() {}

    __device__ float operator () (const uint idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist;
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

template<class Int, class Scalar>
void cpuTestFixedPoint
(
        Scalar *p
        )
{
    Int q[64];
    int emax = fwd_cast(q, p, 1,nx,nx*ny);
}

int main()
{
    device_vector<double> d_vec_in(nx*ny*nz), d_vec_out(nx*ny*nz);
    host_vector<double> h_vec_in(nx*ny*nz);

    thrust::counting_iterator<uint> index_sequence_begin(0);
    thrust::transform(
                    index_sequence_begin,
                    index_sequence_begin + nx*ny*nz,
                    d_vec_in.begin(),
                    RandGen());

    h_vec_in = d_vec_in;
    cpuTestFixedPoint<long long, double>(raw_pointer_cast(h_vec_in.data()));
}
