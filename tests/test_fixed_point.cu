#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#define KEPLER 0
#include "ErrorCheck.h"
#include "fixed_point.cuh"

using namespace thrust;
using namespace std;

#define FREXP(x, e) frexp(x, e)
#define LDEXP(x, e) ldexp(x, e)

const int nx = 256;
const int ny = 256;
const int nz = 256;
device_vector<double> d_vec_in(nx*ny*nz);
device_vector<long long> d_vec_out(nx*ny*nz);
host_vector<double> h_vec_in(nx*ny*nz);


//Used to generate rand array in CUDA with Thrust
struct RandGen
{
    RandGen() {}

    __device__ float operator () (const uint idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist(0.0, 0.0001);
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

template<class Int, class Scalar>
void cpuTestFixedPoint
(
        Scalar *p
        )
{
#pragma omp parallel for
    for (int z=0; z<nz; z+=4){
        for (int y=0; y<ny; y+=4){
            for (int x=0; x<nx; x+=4){
                int idx = z*nx*ny + y*nx + x;
                Int q[64];
                Int q2[64];
                int emax2 = max_exp<Scalar>(p, idx, 1,nx,nx*ny);
                fixed_point(q2,p, emax2, idx, 1,nx,nx*ny);

                int emax = fwd_cast(q, p+idx, 1,nx,nx*ny);

                for (int i=0; i<64; i++){
                    assert(q[i] == q2[i]);
                }

            }
        }
    }

}
template<class Int, class Scalar>
__global__
void cudaFixedPoint
(
        const int *emax,
        const Scalar *data,
        Int *q
        )
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y  + blockDim.y*blockIdx.y;
    int z = threadIdx.z + blockDim.z*blockIdx.z;
    int eidx = z*gridDim.x*blockDim.x*gridDim.y*blockDim.y + y*gridDim.x*blockDim.x + x;

    x *= 4; y*=4; z*=4;
    int idx = z*gridDim.x*gridDim.y*blockDim.x*blockDim.y*16 + y*gridDim.x*blockDim.x*4+ x;
    fixed_point(q, data, emax[eidx], idx, gridDim.x*blockDim.x*4, gridDim.y*blockDim.y*4, gridDim.z*blockDim.z*4);
}

template<class Scalar>
__global__
void cudaMaxExp
(
        int *emax,
    Scalar *data
        )
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y  + blockDim.y*blockIdx.y;
    int z = threadIdx.z + blockDim.z*blockIdx.z;
    int eidx = z*gridDim.x*blockDim.x*gridDim.y*blockDim.y + y*gridDim.x*blockDim.x + x;

    x *= 4; y*=4; z*=4;
    int idx = z*gridDim.x*gridDim.y*blockDim.x*blockDim.y*16 + y*gridDim.x*blockDim.x*4+ x;
    emax[eidx] = max_exp(data, idx, gridDim.x*blockDim.x*4, gridDim.y*blockDim.y*4, gridDim.z*blockDim.z*4);

}

template<class Int, class Scalar>
void gpuTestFixedPoint
(
        device_vector<Scalar> &data,
        device_vector<Int> &q,
        device_vector<int> &emax
        )
{
    dim3 emax_size(nx/4, ny/4, nz/4 );

    dim3 block_size(8,8,8);
    dim3 grid_size = emax_size;
    grid_size.x /= block_size.x; grid_size.y /= block_size.y;  grid_size.z /= block_size.z;

    ErrorCheck ec;

    ec.chk("pre-cudaMaxExp");
    cudaMaxExp<<<block_size,grid_size>>>
            (
                raw_pointer_cast(emax.data()),
                raw_pointer_cast(data.data())
                );
    ec.chk("cudaMaxExp");

    ec.chk("pre-cudaFixedPoint");
    cudaFixedPoint<<<block_size, grid_size>>>
            (
                raw_pointer_cast(emax.data()),
                raw_pointer_cast(data.data()),
                raw_pointer_cast(q.data())
                );
    ec.chk("cudaFixedPoint");
    host_vector<int> h_emax;
    host_vector<Scalar> h_p;
    host_vector<Int> h_q;
    h_emax = emax;
    h_p = data;
    h_q = q;

    int i=0;
    for (int z=0; z<nz; z+=4){
        for (int y=0; y<ny; y+=4){
            for (int x=0; x<nx; x+=4){
                int idx = z*nx*ny + y*nx + x;
                Int q2[64];
                int emax2 = max_exp<Scalar>(raw_pointer_cast(h_p.data()), idx, 1,nx,nx*ny);
                assert(emax2 == h_emax[i++]);
                fixed_point(q2,raw_pointer_cast(h_p.data()), emax2, idx, 1,nx,nx*ny);
                for (int j=0; j<64; j++){
                    assert(h_q[j+(i-1)*64] == q2[j]);
                }
            }
        }
    }
}

int main()
{

    dim3 emax_size(nx/4, ny/4, nz/4);
    device_vector<int> emax(emax_size.x * emax_size.y * emax_size.z);

    thrust::counting_iterator<uint> index_sequence_begin(0);
    thrust::transform(
                    index_sequence_begin,
                    index_sequence_begin + nx*ny*nz,
                    d_vec_in.begin(),
                    RandGen());

    gpuTestFixedPoint<long long, double>(d_vec_in, d_vec_out, emax);
    h_vec_in = d_vec_in;
    cpuTestFixedPoint<long long, double>(raw_pointer_cast(h_vec_in.data()));
}
