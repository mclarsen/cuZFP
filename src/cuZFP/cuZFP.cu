#include "hip/hip_runtime.h"
#include <assert.h>
#include "cuZFP.h"
#include "encode.cuh"
#include "encode1.cuh"
#include "ErrorCheck.h"
#include "decode.cuh"
#include <constant_setup.cuh>
#include <thrust/device_vector.h>
#include <iostream>

namespace cuZFP {
namespace internal {

template<typename T>
void encode(int nx, int ny, int nz, std::vector<T> &in_data, EncodedData &encoded_data)
{

  ErrorCheck errors;
   
  int3 dims = make_int3(nx, ny, nz);
  const int bsize = encoded_data.m_bsize;

  assert(in_data.size() == nx * ny * nz);
  // device mem where encoded data is stored
  // allocate in encode
  thrust::device_vector<Word> d_encoded;
  thrust::device_vector<T> d_in_data(in_data); 

  ConstantSetup::setup_3d();

  cuZFP::encode<T>(dims, d_in_data, d_encoded, bsize); 

  errors.chk("Encode");
  encoded_data.m_data.resize(d_encoded.size());

  Word * d_ptr = thrust::raw_pointer_cast(d_encoded.data());
  Word * h_ptr = &encoded_data.m_data[0];

  // copy the decoded data back to the host
  hipMemcpy(h_ptr, d_ptr, d_encoded.size() * sizeof(Word), hipMemcpyDeviceToHost);

  // set the actual dims and padded dims
  encoded_data.m_dims[0] = nx;
  encoded_data.m_dims[1] = ny;
  encoded_data.m_dims[2] = nz;
}

template<typename T>
void encode(int nx, std::vector<T> &in_data, EncodedData &encoded_data)
{

  ErrorCheck errors;
   
  int dim = nx;
  const int bsize = encoded_data.m_bsize;

  assert(in_data.size() == nx);
  // device mem where encoded data is stored
  // allocate in encode
  thrust::device_vector<Word> d_encoded;
  thrust::device_vector<T> d_in_data(in_data); 
  
  std::cout<<"setting up constants\n";
  ConstantSetup::setup_1d();

  std::cout<<"calling encode\n";
  cuZFP::encode1<T>(dim, d_in_data, d_encoded, bsize); 

  errors.chk("encode1");
  encoded_data.m_data.resize(d_encoded.size());

  Word * d_ptr = thrust::raw_pointer_cast(d_encoded.data());
  Word * h_ptr = &encoded_data.m_data[0];

  // copy the decoded data back to the host
  hipMemcpy(h_ptr, d_ptr, d_encoded.size() * sizeof(Word), hipMemcpyDeviceToHost);

  // set the actual dims and padded dims
  encoded_data.m_dims[0] = nx;
  encoded_data.m_dims[1] = 0;
  encoded_data.m_dims[2] = 0;
}

template<typename T>
void decode(const EncodedData &encoded_data, std::vector<T> &out_data)
{

  const unsigned int bsize = encoded_data.m_bsize;

  int3 dims = make_int3(encoded_data.m_dims[0],
                        encoded_data.m_dims[1],
                        encoded_data.m_dims[2]);

  const size_t out_size = dims.x * dims.y * dims.z;

  thrust::device_vector<T> d_out_data(out_size); 
  thrust::device_vector<Word> d_encoded(encoded_data.m_data);

  ConstantSetup::setup_3d();

  cuZFP::decode<T>(dims, d_encoded, d_out_data, bsize); 

  out_data.resize(out_size); 
  thrust::copy(d_out_data.begin(), 
               d_out_data.end(),
               out_data.begin());
}

} // namespace internal

// 3D encoding
void encode(int nx, int ny, int nz, std::vector<double> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f64;
}

void encode(int nx, int ny, int nz, std::vector<float> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f32;
}

void encode(int nx, int ny, int nz, std::vector<int> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::i32;
}

void encode(int nx, int ny, int nz, std::vector<long long int> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::i64;
}

// 1D encoding
void encode(int nx, std::vector<float> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f32;
}

// 3D decoding
void decode(const EncodedData &encoded_data, std::vector<double> &out_data)
{
  assert(encoded_data.m_value_type = EncodedData::f64);
  internal::decode(encoded_data, out_data);
}

void decode(const EncodedData &encoded_data, std::vector<float> &out_data)
{
  assert(encoded_data.m_value_type = EncodedData::f32);
  internal::decode(encoded_data, out_data);
}

void decode(const EncodedData &encoded_data, std::vector<int> &out_data)
{
  assert(encoded_data.m_value_type = EncodedData::i32);
  internal::decode(encoded_data, out_data);
}

void decode(const EncodedData &encoded_data, std::vector<long long int> &out_data)
{
  assert(encoded_data.m_value_type = EncodedData::i64);
  internal::decode(encoded_data, out_data);
}


} // namespace cuZFP

