#include "hip/hip_runtime.h"
#include <assert.h>
#include "cuZFP.h"
#include "encode.cuh"
#include "encode1.cuh"
#include "encode2.cuh"
#include "ErrorCheck.h"
#include "decode.cuh"
#include "decode1.cuh"
#include "decode2.cuh"
#include <constant_setup.cuh>
#include <thrust/device_vector.h>
#include <iostream>

namespace cuZFP {
namespace internal {

template<typename T>
void encode(int nx, int ny, int nz, std::vector<T> &in_data, EncodedData &encoded_data)
{

  ErrorCheck errors;
   
  int3 dims = make_int3(nx, ny, nz);
  const int bsize = encoded_data.m_bsize;

  assert(in_data.size() == nx * ny * nz);
  // device mem where encoded data is stored
  // allocate in encode
  thrust::device_vector<Word> d_encoded;
  thrust::device_vector<T> d_in_data(in_data); 

  ConstantSetup::setup_3d();

  cuZFP::encode<T>(dims, d_in_data, d_encoded, bsize); 

  errors.chk("Encode");
  encoded_data.m_data.resize(d_encoded.size());

  Word * d_ptr = thrust::raw_pointer_cast(d_encoded.data());
  Word * h_ptr = &encoded_data.m_data[0];

  // copy the decoded data back to the host
  hipMemcpy(h_ptr, d_ptr, d_encoded.size() * sizeof(Word), hipMemcpyDeviceToHost);

  // set the actual dims and padded dims
  encoded_data.m_dims[0] = nx;
  encoded_data.m_dims[1] = ny;
  encoded_data.m_dims[2] = nz;
}

template<typename T>
void encode(int nx, std::vector<T> &in_data, EncodedData &encoded_data)
{

  ErrorCheck errors;
   
  int dim = nx;
  const int bsize = encoded_data.m_bsize;

  assert(in_data.size() == nx);
  // device mem where encoded data is stored
  // allocate in encode
  thrust::device_vector<Word> d_encoded;
  thrust::device_vector<T> d_in_data(in_data); 
  
  ConstantSetup::setup_1d();

  cuZFP::encode1<T>(dim, d_in_data, d_encoded, bsize); 

  errors.chk("encode1");
  encoded_data.m_data.resize(d_encoded.size());

  Word * d_ptr = thrust::raw_pointer_cast(d_encoded.data());
  Word * h_ptr = &encoded_data.m_data[0];

  // copy the decoded data back to the host
  hipMemcpy(h_ptr, d_ptr, d_encoded.size() * sizeof(Word), hipMemcpyDeviceToHost);

  // set the actual dims and padded dims
  encoded_data.m_dims[0] = nx;
  encoded_data.m_dims[1] = 0;
  encoded_data.m_dims[2] = 0;
}

template<typename T>
void encode(int nx, int ny, std::vector<T> &in_data, EncodedData &encoded_data)
{

  ErrorCheck errors;
   
  int2 dims;
  dims.x = nx;
  dims.y = ny;
  const int bsize = encoded_data.m_bsize;

  assert(in_data.size() == nx * ny);
  // device mem where encoded data is stored
  // allocate in encode
  thrust::device_vector<Word> d_encoded;
  thrust::device_vector<T> d_in_data(in_data); 
  
  std::cout<<"setting up constants\n";
  ConstantSetup::setup_2d();

  std::cout<<"calling encode\n";
  cuZFP::encode2<T>(dims, d_in_data, d_encoded, bsize); 

  errors.chk("encode2");
  encoded_data.m_data.resize(d_encoded.size());

  Word * d_ptr = thrust::raw_pointer_cast(d_encoded.data());
  Word * h_ptr = &encoded_data.m_data[0];

  // copy the decoded data back to the host
  hipMemcpy(h_ptr, d_ptr, d_encoded.size() * sizeof(Word), hipMemcpyDeviceToHost);

  // set the actual dims and padded dims
  encoded_data.m_dims[0] = nx;
  encoded_data.m_dims[1] = ny;
  encoded_data.m_dims[2] = 0;
}

template<typename T>
void decode(const EncodedData &encoded_data, std::vector<T> &out_data)
{

  const unsigned int bsize = encoded_data.m_bsize;

  int d = 0;
  for(int i = 0; i < 3; ++i)
  {
    if(encoded_data.m_dims[i] != 0) d++;
  }
  if(d == 3)
  {
    int3 dims = make_int3(encoded_data.m_dims[0],
                          encoded_data.m_dims[1],
                          encoded_data.m_dims[2]);
    const size_t out_size = dims.x * dims.y * dims.z;

    thrust::device_vector<T> d_out_data(out_size); 
    thrust::device_vector<Word> d_encoded(encoded_data.m_data);

    ConstantSetup::setup_3d();

    cuZFP::decode<T>(dims, d_encoded, d_out_data, bsize); 

    out_data.resize(out_size); 
    thrust::copy(d_out_data.begin(), 
                 d_out_data.end(),
                 out_data.begin());
  }
  else if(d == 1)
  {

    int dim = encoded_data.m_dims[0];
    const size_t out_size = dim;

    thrust::device_vector<T> d_out_data(out_size); 
    thrust::device_vector<Word> d_encoded(encoded_data.m_data);

    ConstantSetup::setup_1d();

    cuZFP::decode1<T>(dim, d_encoded, d_out_data, bsize); 

    out_data.resize(out_size); 
    thrust::copy(d_out_data.begin(), 
                 d_out_data.end(),
                 out_data.begin());
  }
  else if(d == 2)
  {

    int2 dims;
    dims.x = encoded_data.m_dims[0];
    dims.y = encoded_data.m_dims[1];
    const size_t out_size = dims.x * dims.y;

    thrust::device_vector<T> d_out_data(out_size); 
    thrust::device_vector<Word> d_encoded(encoded_data.m_data);

    ConstantSetup::setup_2d();

    cuZFP::decode2<T>(dims, d_encoded, d_out_data, bsize); 

    out_data.resize(out_size); 
    thrust::copy(d_out_data.begin(), 
                 d_out_data.end(),
                 out_data.begin());
  }
  else std::cout<<" d ==  "<<d<<" not implemented\n";
  
}

} // namespace internal

// -----------------------------  3D encoding -------------------------------------------
void encode(int nx, int ny, int nz, std::vector<double> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f64;
}

void encode(int nx, int ny, int nz, std::vector<float> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f32;
}

void encode(int nx, int ny, int nz, std::vector<int> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::i32;
}

void encode(int nx, int ny, int nz, std::vector<long long int> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, nz, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::i64;
}

// -------------------------- 1D encoding --------------------------------------------
void encode(int nx, std::vector<float> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f32;
}

void encode(int nx, std::vector<double> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f64;
}

void encode(int nx, std::vector<int> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::i32;
}

void encode(int nx, std::vector<long long int> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::i64;
}

// -------------------------- 2D encoding --------------------------------------------
void encode(int nx, int ny, std::vector<float> &in_data, EncodedData &encoded_data)
{
  internal::encode(nx, ny, in_data, encoded_data);  
  encoded_data.m_value_type = EncodedData::f32;
}

// --------------------------- decoding --------------------------------------------
void decode(const EncodedData &encoded_data, std::vector<double> &out_data)
{
  //assert(encoded_data.m_value_type = EncodedData::f64);
  internal::decode(encoded_data, out_data);
}

void decode(const EncodedData &encoded_data, std::vector<float> &out_data)
{
  //assert(encoded_data.m_value_type = EncodedData::f32);
  internal::decode(encoded_data, out_data);
}

void decode(const EncodedData &encoded_data, std::vector<int> &out_data)
{
  //assert(encoded_data.m_value_type = EncodedData::i32);
  internal::decode(encoded_data, out_data);
}

void decode(const EncodedData &encoded_data, std::vector<long long int> &out_data)
{
  //assert(encoded_data.m_value_type = EncodedData::i64);
  internal::decode(encoded_data, out_data);
}


} // namespace cuZFP

