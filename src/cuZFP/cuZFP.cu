#include "hip/hip_runtime.h"
#include <assert.h>
#include "cuZFP.h"

#include "encode1.cuh"
#include "encode2.cuh"
#include "encode3.cuh"

#include "ErrorCheck.h"

#include "decode1.cuh"
#include "decode2.cuh"
#include "decode3.cuh"

#include <constant_setup.cuh>
#include <pointers.cuh>
#include <thrust/device_vector.h>
#include <iostream>
#include <type_info.cuh>

namespace cuZFP {
namespace internal {

//
// encode expects device pointers
//
template<typename T>
size_t encode(int dims[3], int bits_per_block, T *d_data, Word *d_stream)
{

  int d = 0;
  size_t len = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      d++;
      len *= dims[i];
    }
  }

  ErrorCheck errors;
  size_t stream_size = 0;
  if(d == 1)
  {
    int dim = dims[0];
    ConstantSetup::setup_1d();
    stream_size = cuZFP::encode1<T>(dim, d_data, d_stream, bits_per_block); 
  }
  else if(d == 2)
  {
    int2 ndims = make_int2(dims[0], dims[1]);
    ConstantSetup::setup_2d();
    stream_size = cuZFP::encode2<T>(ndims, d_data, d_stream, bits_per_block); 
  }
  else if(d == 3)
  {
    int3 ndims = make_int3(dims[0], dims[1], dims[2]);
    ConstantSetup::setup_3d();
    stream_size = cuZFP::encode<T>(ndims, d_data, d_stream, bits_per_block); 
  }

  errors.chk("Encode");
  return stream_size; 
}

template<typename T>
void decode(int ndims[3], int bits_per_block, Word *stream, T *out)
{

  int d = 0;
  size_t out_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(ndims[i] != 0)
    {
      d++;
      out_size *= ndims[i];
    }
  }

  if(d == 3)
  {
    int3 dims = make_int3(ndims[0], ndims[1], ndims[2]);
    ConstantSetup::setup_3d();
    cuZFP::decode3<T>(dims, stream, out, bits_per_block); 
  }
  else if(d == 1)
  {
    int dim = ndims[0];
    ConstantSetup::setup_1d();
    cuZFP::decode1<T>(dim, stream, out, bits_per_block); 

  }
  else if(d == 2)
  {
    int2 dims;
    dims.x = ndims[0];
    dims.y = ndims[1];
    ConstantSetup::setup_2d();
    cuZFP::decode2<T>(dims, stream, out, bits_per_block); 

  }
  else std::cerr<<" d ==  "<<d<<" not implemented\n";
  
}

Word *setup_device_stream(zfp_stream *stream, zfp_field *field)
{
  bool stream_device = is_gpu_ptr(stream->stream);

  if(stream_device)
  {
    return stream->stream;
  }

  Word *d_stream = NULL;
  // TODO: we we have a real stream we can just ask it how big it is
  size_t max_size = zfp_stream_maximum_size(stream, field);
  hipMalloc(&d_stream, max_size);
  hipMemcpy(d_stream, stream->stream, max_size, hipMemcpyHostToDevice);
  return d_stream;
}

void *setup_device_field(zfp_field *field)
{
  bool field_device = is_gpu_ptr(field->data);

  if(field_device)
  {
    return field->data;
  }
  
  int dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;

  size_t type_size = zfp_type_size(field->type);

  size_t field_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      field_size *= dims[i];
    }
  }

  void *d_data = NULL;

  size_t field_bytes = type_size * field_size;
  hipMalloc(&d_data, field_bytes);
  hipMemcpy(d_data, field->data, field_bytes, hipMemcpyHostToDevice);
  return d_data;
}

void cleanup_device_ptr(void *orig_ptr, void *d_ptr, size_t bytes)
{
  bool device = is_gpu_ptr(orig_ptr);
  if(device)
  {
    return;
  }
  // from whence it came
  if(bytes > 0)
  {
    hipMemcpy(orig_ptr, d_ptr, bytes, hipMemcpyDeviceToHost);
  }
  hipFree(d_ptr);
}

} // namespace internal

size_t
compress(zfp_stream *stream, zfp_field *field)
{
  int dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;
  size_t stream_bytes = 0;
  
  void *d_data = internal::setup_device_field(field);
  Word *d_stream = internal::setup_device_stream(stream, field);

  if(field->type == zfp_type_float)
  {
    float* data = (float*) d_data;
    stream_bytes = internal::encode<float>(dims, (int)stream->maxbits, data, d_stream);
  }
  else if(field->type == zfp_type_double)
  {
    double* data = (double*) d_data;
    stream_bytes = internal::encode<double>(dims, (int)stream->maxbits, data, d_stream);
  }
  else if(field->type == zfp_type_int32)
  {
    int * data = (int*) d_data;
    stream_bytes = internal::encode<int>(dims, (int)stream->maxbits, data, d_stream);
  }
  else if(field->type == zfp_type_int64)
  {
    long long int * data = (long long int*) d_data;
    stream_bytes = internal::encode<long long int>(dims, (int)stream->maxbits, data, d_stream);
  }

  internal::cleanup_device_ptr(stream->stream, d_stream, stream_bytes);
  internal::cleanup_device_ptr(field->data, d_data, 0);
  return stream_bytes;
}
  
void 
decompress(zfp_stream *stream, zfp_field *field)
{
  int dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;
    

  void *d_data = internal::setup_device_field(field);
  Word *d_stream = internal::setup_device_stream(stream, field);

  if(field->type == zfp_type_float)
  {
    float *data = (float*) d_data;
    internal::decode(dims, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_double)
  {
    double *data = (double*) d_data;
    internal::decode(dims, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_int32)
  {
    int *data = (int*) d_data;
    internal::decode(dims, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_int64)
  {
    long long int *data = (long long int*) d_data;
    internal::decode(dims, (int)stream->maxbits, d_stream, data);
    d_data = (void*) data;
  }
  else
  {
    std::cerr<<"Cannot decompress: type unknown\n";
  }

   
  size_t type_size = zfp_type_size(field->type);

  size_t field_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      field_size *= dims[i];
    }
  }
  
  size_t bytes = type_size * field_size;
  internal::cleanup_device_ptr(stream->stream, d_stream,0);
  internal::cleanup_device_ptr(field->data, d_data, bytes);
  
}

} // namespace cuZFP

