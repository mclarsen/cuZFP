#include "ErrorCheck.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

ErrorCheck::ErrorCheck()
{

}

void ErrorCheck::chk(std::string msg)
{
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        cout << msg << " : " << error;
        cout << " " << hipGetErrorString(error) << endl;
    }
}

void ErrorCheck::chk()
{
    chk(str.str());
    str.str("");
}
